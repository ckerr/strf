#include "hip/hip_runtime.h"
#include <stdio.h> // for CUDA's printf

#include "test_utils.hpp"
#include <strf.hpp>
#include <sstream>
#include <iostream>

namespace kernels {

// Note: There are adaptations of, say, std::span for use with CUDA (= I adapted it...).
// But we want to avoid dependency clutter here, so let's just stick to the basics.

__global__ void using_cstr_writer(strf::cstr_writer::result* write_result, char* buffer, std::size_t buffer_size)
{
	strf::basic_cstr_writer<char> sw(buffer, buffer_size);
	write(sw, "Hello");
	write(sw, " world");
	*write_result = sw.finish();
}

__global__ void using_cstr_to(char* buffer, std::size_t buffer_size)
{
	int global_thread_id = threadIdx.x + blockIdx.x * blockDim.x;
	auto printer = strf::to(buffer, buffer_size);
	printer ( "Hello", ' ', "world, from thread ", global_thread_id);
}

__global__ void various_types_with_cstr_writer(char* buffer, std::size_t buffer_size)
{
	strf::cstr_writer writer(buffer, buffer_size);
	auto printer = strf::to(writer);

	printer("Printing a bool: ", bool(true), '\n');
//	printer("Printing a wchar_t: ", wchar_t('a'), '\n');
//	printer("Printing a char16_t: ", char16_t('a'), '\n');
//	printer("Printing a char32_t: ", char32_t('a'), '\n');
	printer("Printing a char: ", static_cast<char>('a'), '\n');
	printer("Printing a unsigned char: ", static_cast<unsigned char>('a'), '\n');
	printer("Printing a short: ", short(-1234), '\n');
	printer("Printing a unsigned short: ", static_cast<unsigned short>(1234), '\n');
	printer("Printing a int: ", int(-12345678), '\n');
	printer("Printing a signed int: ", static_cast<signed int>(12345678), '\n');
	printer("Printing a unsigned int: ", static_cast<unsigned int>(12345678), '\n');
	printer("Printing a long int: ", static_cast<long int>(-12345678900), '\n');
	printer("Printing a signed long int: ", static_cast<signed long int>(-12345678900), '\n');
	printer("Printing a unsigned long int: ", static_cast<unsigned long int>(12345678900), '\n');
	printer("Printing a long long int: ", static_cast<long long int>(-12345678900), '\n');
	printer("Printing a signed long long int: ", static_cast<signed long long int>(-12345678900), '\n');
	printer("Printing a unsigned long long int: ", static_cast<unsigned long long int>(12345678900), '\n');
	printer("Printing a float: ", float(1.234567), '\n');
	printer("Printing a double: ", double(1.2345678901234567), '\n');
	writer.finish();
	printf("Buffer contents: \"%s\"", buffer);
}

__global__ void formatting_functions(char* buffer, std::size_t buffer_size)
{
	strf::cstr_writer writer(buffer, buffer_size);
	auto printer = strf::to(writer);

	printer ("strf::fmt(0) gives ", strf::fmt(0), '\n');
	printer ("strf::fmt(0).hex() gives ", strf::fmt(0).hex(), '\n');
	printer ("strf::fmt(0).bin() gives ", strf::fmt(0).bin(), '\n');
	printer ("strf::left(0, 2, '0') gives ", strf::left(0, 2, '0'), '\n');
	printer ("strf::right(0, 2, '0') gives ", strf::right(0, 2, '0'), '\n');
	printer ("strf::fmt(123) gives ", strf::fmt(123), '\n');
	printer ("strf::fmt(123).hex() gives ", strf::fmt(123).hex(), '\n');
	printer ("strf::fmt(123).bin() gives ", strf::fmt(123).bin(), '\n');
	printer ("strf::left(123, 5, '0') gives ", strf::left(123, 5, '0'), '\n');
	printer ("strf::right(123, 5, '0') gives ", strf::right(123, 5, '0'), '\n');

	writer.finish();
}

} // namespace kernels

// Ugly, no-good error-checking.
#define ensure_cuda_success(ans) { ensure_cuda_success_((ans), __FILE__, __LINE__); }

inline void ensure_cuda_success_(hipError_t status, const char *file, int line, bool abort=true)
{
	TEST_EQ(status, hipSuccess);
	if (abort and (status != hipSuccess)) {
		TEST_ERROR(hipGetErrorString(status));
		exit(test_finish());
	}
}

void test_cstr_writer()
{
	struct args {
		strf::cstr_writer::result write_result;
		char buffer[50];
	};
	const std::size_t buffer_size { std::strlen("Hello world") + 1 }; // Enough for "Hello world" with the trailing '\0'.
	struct args* device_side_args;
	ensure_cuda_success(hipMalloc(&device_side_args, sizeof(struct args)));
	ensure_cuda_success(hipMemset(device_side_args, 0, sizeof(struct args)));

	int threads_per_block { 1 };
	int blocks_in_grid { 1 };
		// We could theoretically have multiple threads in multiple blocks run this, but
		// it shouldn't really matter.
	kernels::using_cstr_writer<<<threads_per_block, blocks_in_grid>>>(
		&(device_side_args->write_result),
		&(device_side_args->buffer[0]),
		buffer_size);
	ensure_cuda_success(hipGetLastError());
	ensure_cuda_success(hipDeviceSynchronize());
	args host_side_args;
	ensure_cuda_success(hipMemcpy(&host_side_args, device_side_args, sizeof(struct args), hipMemcpyDeviceToHost));
	TEST_EQ(host_side_args.write_result.truncated, false);
	TEST_EQ(host_side_args.write_result.ptr, &(device_side_args->buffer[0]) + std::strlen("Hello world"));
	if (host_side_args.write_result.ptr == &(device_side_args->buffer[0])) {
		TEST_EQ(strncmp(host_side_args.write_result.ptr, host_side_args.buffer, buffer_size), 0);
	}
}


void test_cstr_to()
{
	char* device_side_buffer;
	const std::size_t buffer_size { 100 }; // More than enough for "Hello world from thread XYZ"
	ensure_cuda_success( hipMalloc(&device_side_buffer, buffer_size) );
	ensure_cuda_success( hipMemset(device_side_buffer, 0, buffer_size) );

	int threads_per_block { 1 };
	int blocks_in_grid { 1 };
		// We could theoretically have multiple threads in multiple blocks run this, but
		// it shouldn't really matter.
	kernels::using_cstr_to<<<threads_per_block, blocks_in_grid>>>(device_side_buffer, buffer_size);
	ensure_cuda_success(hipGetLastError());
	ensure_cuda_success(hipDeviceSynchronize());
	char host_side_buffer[buffer_size];
	ensure_cuda_success(hipMemcpy(&host_side_buffer, device_side_buffer, buffer_size , hipMemcpyDeviceToHost));
	std::stringstream expected;
	expected << "Hello" << ' ' << "world, from thread " << 0;
	TEST_EQ(strncmp(host_side_buffer, expected.str().c_str(), buffer_size), 0);
	std::cout << std::endl;
	std::cout << "Result: \"" << host_side_buffer << "\"\n";
	std::cout << "Expected: \"" << expected.str() <<  "\"\n";
}

void test_various_types_with_cstr_writer()
{
	char* device_side_buffer;
	const std::size_t buffer_size { 5000 }; 
	ensure_cuda_success( hipMalloc(&device_side_buffer, buffer_size) );
	ensure_cuda_success( hipMemset(device_side_buffer, 0, buffer_size) );

	int threads_per_block { 1 };
	int blocks_in_grid { 1 };
		// We could theoretically have multiple threads in multiple blocks run this, but
		// it shouldn't really matter.
	kernels::various_types_with_cstr_writer<<<threads_per_block, blocks_in_grid>>>(device_side_buffer, buffer_size);
	ensure_cuda_success(hipGetLastError());
	ensure_cuda_success(hipDeviceSynchronize());
	char host_side_buffer[buffer_size];
	ensure_cuda_success(hipMemcpy(&host_side_buffer, device_side_buffer, buffer_size , hipMemcpyDeviceToHost));
	std::stringstream expected;
	expected
		<< "Printing a bool: " << bool(true) << '\n'
		<< "Printing a char: " << char('a') << '\n'
		<< "Printing a unsigned char: " << static_cast<unsigned char>('a') << '\n'
		<< "Printing a short: " << short(-1234) << '\n'
		<< "Printing a unsigned short: " << static_cast<unsigned short>(1234) << '\n'
		<< "Printing a int: " << int(-12345678) << '\n'
		<< "Printing a signed int: " << static_cast<signed int>(12345678) << '\n'
		<< "Printing a unsigned int: " << static_cast<unsigned int>(12345678) << '\n'
		<< "Printing a long int: " << static_cast<long int>(-12345678900) << '\n'
		<< "Printing a signed long int: " << static_cast<signed long int>(-12345678900) << '\n'
		<< "Printing a unsigned long int: " << static_cast<unsigned long int>(12345678900) << '\n'
		<< "Printing a long long int: " << static_cast<long long int>(-12345678900) << '\n'
		<< "Printing a signed long long int: " << static_cast<signed long long int>(-12345678900) << '\n'
		<< "Printing a unsigned long long int: " << static_cast<unsigned long long int>(12345678900) << '\n'
		<< "Printing a float: " << float(1.234567) << '\n'
		<< "Printing a double: " << double(1.2345678901234567), '\n';
	TEST_EQ(strncmp(host_side_buffer, expected.str().c_str(), buffer_size), 0);
	std::cout << std::endl;
	std::cout << "Result: \"" << host_side_buffer << "\"\n";
	std::cout << "Expected: \"" << expected.str() <<  "\"\n";
}

void test_formatting_functions()
{
	char* device_side_buffer;
	constexpr std::size_t buffer_size { 400 };
	ensure_cuda_success( hipMalloc(&device_side_buffer, buffer_size) );
	ensure_cuda_success( hipMemset(device_side_buffer, 0, buffer_size) );

	int threads_per_block { 1 };
	int blocks_in_grid { 1 };
	kernels::formatting_functions<<<threads_per_block, blocks_in_grid>>>(device_side_buffer, buffer_size);
	ensure_cuda_success(hipGetLastError());
	ensure_cuda_success(hipDeviceSynchronize());
	char host_side_buffer[buffer_size];
	ensure_cuda_success(hipMemcpy(&host_side_buffer, device_side_buffer, buffer_size , hipMemcpyDeviceToHost));
	std::stringstream expected;
	expected  <<
		"strf::fmt(0) gives 0\n"
		"strf::fmt(0).hex() gives 0\n"
		"strf::fmt(0).bin() gives 0\n"
		"strf::left(0, 2, '0') gives 00\n"
		"strf::right(0, 2, '0') gives 00\n"
		"strf::fmt(123) gives 123\n"
		"strf::fmt(123).hex() gives 7b\n"
		"strf::fmt(123).bin() gives 1111011\n"
		"strf::left(123, 5, '0') gives 12300\n"
		"strf::right(123, 5, '0') gives 00123\n";
	TEST_EQ(strncmp(host_side_buffer, expected.str().c_str(), buffer_size), 0);
	std::cout << std::endl;
	std::cout << "Result: \"" << host_side_buffer << "\"\n";
	std::cout << "Expected: \"" << expected.str() <<  "\"\n";
}


void cstr_to_sanity_check()
{
	const std::size_t buffer_size { 100 }; // More than enough for "Hello world from thread XYZ"
	char buffer[buffer_size];
	std::fill_n(buffer, sizeof(buffer), 0);
	auto print_functor = strf::to(buffer, buffer_size);
	print_functor ( "Hello", ' ', "world, from thread ", 1 );
	std::stringstream expected;
	expected << "Hello" << ' ' << "world, from thread " << 1;
	TEST_EQ(strncmp(buffer, expected.str().c_str(), buffer_size), 0);
}


int main(void)
{
	auto num_devices { 0 };
	auto status = hipGetDeviceCount(&num_devices);

	TEST_EQ(status, hipSuccess);
	if (status != hipSuccess)
	{
		std::stringstream ss;
		ss << "hipGetDeviceCount failed: " << hipGetErrorString(status) <<  '\n';
		TEST_ERROR(ss.str().c_str());
	}
	if (num_devices == 0) {
		std::cerr << "No devices - can't run this test\n";
		return test_finish();
	}
	// TODO: Test basic_cstr_writer's with different character types
	test_cstr_writer();
	cstr_to_sanity_check();
	test_cstr_to();
	test_formatting_functions();
	test_various_types_with_cstr_writer();

	hipDeviceReset();
	return test_finish();
}
